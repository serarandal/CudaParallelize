


#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define anchoBloque 16  
//-------------------------------------------------------------------
void initMatriz (float *M, int card, float valor) {
  int i;
  for (i=0; i<card; i++) M[i] = valor;
}

//-------------------------------------------------------------------
__global__ void mulMatrizKernel (float *Ad, float *Bd, float *Cd, int card) {
  int ROW= blockIdx.x*blockDim.x+threadIdx.x;
  int COL= blockIdx.y*blockDim.y+threadIdx.y;
  float tmpSum=0;
 
  if(ROW < card && COL < card){ 
for (int i = 0; i<card; i++){
   tmpSum = tmpSum + Ad[ROW*card+i] * Bd[i*card + COL]; }
}
 Cd[ROW*card+COL] = tmpSum;
}
//-------------------------------------------------------------------
int main (int argc, char *argv[])
{
  int filA, colA, filB, colB, filC, colC;
  struct timeval t0, tf, t;
  float  *A, *B, *C;
  float  *Ad, *Bd, *Cd;
  int    sizeA, sizeB, sizeC, k;

  filA = atoi(argv[1]);
  colA = filA;
  filB = filA;
  colB = filA;
  filC = filA;
  colC = filA;
  sizeA = filA*colA*sizeof(float);
  sizeB = filB*colB*sizeof(float);
  sizeC = filC*colC*sizeof(float);
  A = (float *) malloc (sizeA);
  B = (float *) malloc (sizeB);
  C = (float *) malloc (sizeC);
  initMatriz (A, filA*colA, 1.0f );
  initMatriz (B, filB*colB, 0.01f);

  assert (gettimeofday (&t0, NULL) == 0);
  
  hipMalloc ((void**) &Ad, sizeA);
  hipMemcpy (Ad, A, sizeA, hipMemcpyHostToDevice);
  hipMalloc ((void**) &Bd, sizeB);
  hipMemcpy (Bd, B, sizeB, hipMemcpyHostToDevice);
  
  hipMalloc ((void**) &Cd, sizeC);
  
  dim3 dimGrid (filA/anchoBloque, filA/anchoBloque);
  dim3 dimBlock(anchoBloque, anchoBloque);
  mulMatrizKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, filA);
  hipDeviceSynchronize();
 
  hipMemcpy (C, Cd, sizeC, hipMemcpyDeviceToHost);
  
  hipFree(Ad); hipFree(Bd); hipFree(Cd);
  assert (gettimeofday (&tf, NULL) == 0);

  timersub (&tf, &t0, &t);
  printf ("Tiempo = %ld:%ld \n", t.tv_sec, t.tv_usec);
  printf ("C[0] = %f\n", C[filC-1]);
  for (k=1; k<(filC*colC); k++) assert (C[k] == C[k-1]);
  return 0;
}
